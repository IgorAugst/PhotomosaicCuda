#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "preProcess.h"

#include <stdio.h>
#include <iostream>
#include <math.h>

#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/highgui.hpp>
#include <set>

using namespace std;
using namespace cv;

#define MAX 32

typedef struct RGB {
	int r;
	int g;
	int b;
};

typedef struct imageBlock {
	char name[MAXNAME];
};

__global__ void AvrgKernel(unsigned char* input, int colorWidthStep, int imgWidth, int imgHeight, ImageData* imData, int imgQuant, ImageData* out) {
	int blockX = blockIdx.x;
	int blockY = blockIdx.y;

	int blockWidth = imgWidth / gridDim.x;
	int blockHeight = imgHeight / gridDim.y;

	int sx = blockX * blockWidth;
	int sy = blockY * blockHeight;

	int fx = sx + blockWidth;
	int fy = sy + blockHeight;

	if (fx > imgWidth) {
		fx = imgWidth;
	}

	if (fy > imgHeight) {
		fy = imgHeight;
	}
	
	RGB color;
	color.r = 0;
	color.g = 0;
	color.b = 0;

	for (int y = sy; y < fy; y++) {
		for (int x = sx; x < fx; x++) {
			int index = y * colorWidthStep + (3 * x);

			color.b += input[index];
			color.g += input[index + 1];
			color.r += input[index + 2];
		}
	}

	int n = blockWidth * blockHeight;

	color.b /= n;
	color.g /= n;
	color.r /= n;

	printf("R: %d, G: %d, B: %d\n", color.r, color.g, color.b);

	RGB diff;
	diff.r = color.r - imData[0].R;
	diff.g = color.g - imData[0].G;
	diff.b = color.b - imData[0].B;

	int lowDist = diff.r * diff.r + diff.g * diff.g + diff.b * diff.b;
	int lowIndex = 0;

	for (int i = 1; i < imgQuant; i++) {
		diff.r = color.r - imData[i].R;
		diff.g = color.g - imData[i].G;
		diff.b = color.b - imData[i].B;

		int distAux = diff.r * diff.r + diff.g * diff.g + diff.b * diff.b;

		if (distAux < lowDist) {
			lowDist = distAux;
			lowIndex = i;
		}
	}

	int gridWidth = gridDim.x;

	int index = blockY * gridWidth + blockX;

	
	out[index] = imData[lowIndex];

	printf("%s\n", imData[lowIndex].name);
}

__global__ void FillImageKernel(unsigned char* output, int outputStep, dim3 outputSize, ImageData* imData, dim3 quantBlock, dim3 blockImgSize, unsigned char* blockImg, int blockStep, int hex) {
	
	int outX = blockDim.x * blockIdx.x + threadIdx.x;
	int outY = blockDim.y * blockIdx.y + threadIdx.y;

	dim3 outputTotalSize(outputSize.x * quantBlock.x, outputSize.y * quantBlock.y);

	if (outX > outputTotalSize.x || outY > outputTotalSize.y) {
		return;
	}

	int blockImgX = outX / (outputSize.x / quantBlock.x);
	int blockImgY = outY / (outputSize.y / quantBlock.y);

	int blockImgIndex = blockImgY * quantBlock.x + blockImgX;

	if (imData[blockImgIndex].hex != hex) {
		return;
	}

	float rX = (float)outputSize.x / (float)blockImgSize.x;
	float rY = (float)outputSize.y / (float)blockImgSize.y;

	int pixSubX = outX / rX;
	int pixSubY = outY / rY;

	int outputIndex = outY * outputStep + (3 * outX);
	int subIndex = pixSubY * blockStep + (3 * pixSubX);

	output[outputIndex] = blockImg[subIndex];
	output[outputIndex + 1] = blockImg[subIndex + 1];
	output[outputIndex + 2] = blockImg[subIndex + 2];

}

void cacheTest() {
	ImageList* imlist = processImage("D:\\igora\\Documents\\Code\\Photomosaic\\testes");

	bool status = saveCache(imlist);

	cout << (status ? "salvo" : "erro") << endl;

	free(imlist->image);
	free(imlist);

	imlist = readCache();

	cout << (imlist != NULL ? "lido" : "erro") << endl;

	getchar();
}

/*
void averageTest() {
	Mat image = imread("D:\\igora\\Pictures\\teste.png");
	namedWindow("teste");
	imshow("teste", image);

	unsigned char *dImage;
	int size = image.step * image.rows;

	hipMalloc<unsigned char>(&dImage, size);
	hipMemcpy(dImage, image.ptr(), size, hipMemcpyHostToDevice);

	dim3 blocks(image.rows);
	
	RGB* values;

	hipMallocManaged(&values, sizeof(RGB) * image.rows);

	grayAvrgTestLine<<<blocks, 1>>>(dImage, image.step, image.cols, values);

	hipDeviceSynchronize();

	grayAvrgTestRow << <1, 1 >> > (values, image.rows);

	hipDeviceSynchronize();

	printf("R: %d, G: %d, B: %d", values[0].r, values[0].g, values[0].b);

	printf("\nteste loop\n");

	AvrgKernel<<<2,1>>>(dImage, image.step, image.cols, image.rows);

	hipDeviceSynchronize();

	waitKey(0);

}
*/

void bestImageTest() {
	Mat image = imread("D:\\igora\\Pictures\\quad.png");
	unsigned char* dImage;

	int size = image.rows * image.step;

	hipMalloc<unsigned char>(&dImage, size);
	hipMemcpy(dImage, image.ptr(), size, hipMemcpyHostToDevice);

	dim3 block(2, 2);
	
	int blockWidth = image.cols / block.x;
	int blockHeight = image.rows / block.y;

	ImageList* imgList = processImage("D:\\igora\\Documents\\Code\\Photomosaic\\testes");
	ImageData* imgData;

	hipMalloc<ImageData>(&imgData, sizeof(ImageData) * imgList->n);
	hipMemcpy(imgData, imgList->image, sizeof(ImageData) * imgList->n, hipMemcpyHostToDevice);

	ImageData* out;
	ImageData* outDevData;

	hipMalloc<ImageData>(&outDevData, sizeof(ImageData) * block.x * block.y);

	out = (ImageData*)malloc(sizeof(ImageData) * block.x * block.y);

	AvrgKernel<<<block, 1>>>(dImage, image.step, image.cols, image.rows, imgData, imgList->n, outDevData);

	hipDeviceSynchronize();

	hipMemcpy(out, outDevData, sizeof(ImageData) * block.x * block.y, hipMemcpyDeviceToHost);

	//------------------------------------------------------
	//Leitura das imagens e distribuição

	int quantBlock = block.x * block.y;
	unsigned char** imgArray;
	imgArray = (unsigned char**)malloc(sizeof(char*) * quantBlock);
	
	set<int> usedImg;

	dim3 outSize(500, 500);

	dim3 outImageSize(outSize.x * block.x, outSize.y * block.y);

	Mat outImg(outImageSize.x, outImageSize.y, CV_8UC3);
	unsigned char* outDev;
	int outImgSize = outImg.step * outImg.rows;

	hipMalloc<unsigned char>(&outDev, outImgSize);

	for (int i = 0; i < quantBlock; i++) {
		if (usedImg.find(out[i].hex) == usedImg.end()) {
			Mat imgAux = imread(out[i].name);
			int size = imgAux.rows * imgAux.step;

			hipMalloc<unsigned char>(&imgArray[i], size);
			hipMemcpy(imgArray[i], imgAux.ptr(), size, hipMemcpyHostToDevice);

			dim3 blockImgSize(imgAux.cols, imgAux.rows);

			dim3 threads(MAX, MAX);

			dim3 blockKernel(outImageSize.x / MAX, outImageSize.y / MAX);

			blockKernel.x++;
			blockKernel.y++;

			FillImageKernel<<<blockKernel, threads>>>(outDev, outImg.step, outSize, outDevData, block, blockImgSize, imgArray[i], imgAux.step, out[i].hex);

			usedImg.insert(out[i].hex);
		}

		//TODO: utilizar hashmap para não ler a mesma imagem mais de uma vez
		//TODO: kernel para preencher a imagem
		//TODO: definir o tamanho da imagem
		//TODO: alocar imagem final na gpu
		//TODO: realizar calculo para encontrar os pixels correspondentes quando cortar a imagem (cortar pelo canto ou centro)
			
	}


	hipDeviceSynchronize();

	hipMemcpy(outImg.ptr(), outDev, outImgSize, hipMemcpyDeviceToHost);

	namedWindow("vai");
	imshow("vai", outImg);

	waitKey();
}

int main(int argc, char** argv) {

	//cacheTest();
	//averageTest();
	bestImageTest();

}