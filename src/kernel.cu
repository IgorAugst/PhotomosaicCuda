#include "hip/hip_runtime.h"
﻿#include "kernel.h"

#define MAX 32

typedef struct RGB {
	int r;
	int g;
	int b;
};

typedef struct imageBlock {
	char name[MAXNAME];
};

__global__ void AvrgKernel(unsigned char* input, int colorWidthStep, int imgWidth, int imgHeight, ImageData* imData, int imgQuant, ImageData* out) {
	int blockX = blockIdx.x;
	int blockY = blockIdx.y;

	float blockWidth = (float)imgWidth / (float)gridDim.x;
	float blockHeight = (float)imgHeight / (float)gridDim.y;

	int sx = blockX * blockWidth;
	int sy = blockY * blockHeight;

	int fx = sx + blockWidth;
	int fy = sy + blockHeight;

	if (fx > imgWidth) {
		fx = imgWidth;
	}

	if (fy > imgHeight) {
		fy = imgHeight;
	}
	
	RGB color;
	color.r = 0;
	color.g = 0;
	color.b = 0;

	for (int y = sy; y < fy; y++) {
		for (int x = sx; x < fx; x++) {
			int index = y * colorWidthStep + (3 * x);

			color.b += input[index];
			color.g += input[index + 1];
			color.r += input[index + 2];
		}
	}

	int n = blockWidth * blockHeight;

	color.b /= n;
	color.g /= n;
	color.r /= n;

	RGB diff;
	diff.r = color.r - imData[0].R;
	diff.g = color.g - imData[0].G;
	diff.b = color.b - imData[0].B;

	int lowDist = diff.r * diff.r + diff.g * diff.g + diff.b * diff.b;
	int lowIndex = 0;

	for (int i = 1; i < imgQuant; i++) {
		diff.r = color.r - imData[i].R;
		diff.g = color.g - imData[i].G;
		diff.b = color.b - imData[i].B;

		int distAux = diff.r * diff.r + diff.g * diff.g + diff.b * diff.b;

		if (distAux < lowDist) {
			lowDist = distAux;
			lowIndex = i;
		}
	}

	int gridWidth = gridDim.x;

	int index = blockY * gridWidth + blockX;

	
	out[index] = imData[lowIndex];

	//printf("%s\n", out[index].name);
}

__global__ void FillImageKernel(unsigned char* output, int outputStep, dim3 outputSize, ImageData* imData, dim3 quantBlock, dim3 blockImgSize, unsigned char* blockImg, int blockStep, int hex) {
	
	int outX = blockDim.x * blockIdx.x + threadIdx.x;
	int outY = blockDim.y * blockIdx.y + threadIdx.y;

	dim3 outputPartialSize(outputSize.x * quantBlock.x, outputSize.y * quantBlock.y);

	if (outX >= outputPartialSize.x || outY >= outputPartialSize.y) {
		return;
	}

	int outputTotalSize = outputPartialSize.x * outputPartialSize.y;

	int outputIndex = outY * outputStep + (3 * outX);

	//printf("pixel %d\n", outputIndex);

	int blockImgY = (float)outY / (float)outputSize.y;
	int blockImgX = (float)outX / (float)outputSize.x;

	int blockImgIndex = blockImgY * quantBlock.x + blockImgX;

	
	//printf("%d %d\n", outX, outY);

	if (imData[blockImgIndex].hex != hex) {
		return;
	}

	//printf("i: %d\n", blockImgIndex);

	float rX = (float)outputSize.x / ((float)blockImgSize.x);
	float rY = (float)outputSize.y / ((float)blockImgSize.y);

	int pixSubX = (outX % outputSize.x) / rX;
	int pixSubY = (outY % outputSize.y) / rY;

	
	int subIndex = pixSubY * blockStep + (3 * pixSubX);

	output[outputIndex] = blockImg[subIndex];
	output[outputIndex + 1] = blockImg[subIndex + 1];
	output[outputIndex + 2] = blockImg[subIndex + 2];

}

__global__ void ToGrayScaleKernel(unsigned char* input, int inputStep, dim3 imageDim) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= imageDim.x || y >= imageDim.y) {
		return;
	}

	int index = y * inputStep + (3 * x);

	int gray = 0;

	gray = (input[index] * 0.11) + (input[index + 1] * 0.59 )+ (input[index + 2] * 0.3);

	input[index] = gray;
	input[index + 1] = gray;
	input[index + 2] = gray;
}

/*
void averageTest() {
	Mat image = imread("D:\\igora\\Pictures\\teste.png");
	namedWindow("teste");
	imshow("teste", image);

	unsigned char *dImage;
	int size = image.step * image.rows;

	hipMalloc<unsigned char>(&dImage, size);
	hipMemcpy(dImage, image.ptr(), size, hipMemcpyHostToDevice);

	dim3 blocks(image.rows);
	
	RGB* values;

	hipMallocManaged(&values, sizeof(RGB) * image.rows);

	grayAvrgTestLine<<<blocks, 1>>>(dImage, image.step, image.cols, values);

	hipDeviceSynchronize();

	grayAvrgTestRow << <1, 1 >> > (values, image.rows);

	hipDeviceSynchronize();

	printf("R: %d, G: %d, B: %d", values[0].r, values[0].g, values[0].b);

	printf("\nteste loop\n");

	AvrgKernel<<<2,1>>>(dImage, image.step, image.cols, image.rows);

	hipDeviceSynchronize();

	waitKey(0);

}
*/

ImageList* Average(Mat img, ImageList* imList, int x) {
	float ratio = (float)img.cols / (float)img.rows;
	int yBlocks = ceil(x / ratio);
	dim3 blockKernel(x, yBlocks);

	unsigned char* dImage;
	int size = img.rows * img.step;

	hipMalloc<unsigned char>(&dImage, size);
	hipMemcpy(dImage, img.ptr(), size, hipMemcpyHostToDevice); //aloca e copia a imagem para gpu

	ImageData* imData;
	hipMalloc<ImageData>(&imData, sizeof(ImageData) * imList->n);
	hipMemcpy(imData, imList->image, sizeof(ImageData) * imList->n, hipMemcpyHostToDevice); //aloca e copia o cache das imagens

	ImageData* outData;
	hipMalloc<ImageData>(&outData, sizeof(ImageData) * x * yBlocks); //aloca os dados de saida

	AvrgKernel<<<blockKernel, 1>>>(dImage, img.step, img.cols, img.rows, imData, imList->n, outData);

	hipDeviceSynchronize();

	ImageData* hostData;
	hostData = (ImageData*)malloc(sizeof(ImageData) * x * yBlocks);

	hipMemcpy(hostData, outData, sizeof(ImageData) * x * yBlocks, hipMemcpyDeviceToHost); // copia os dados para o host

	hipFree(outData);
	hipFree(imData);
	hipFree(dImage);

	ImageList* outList = (ImageList*)malloc(sizeof(ImageList));
	outList->image = hostData;
	outList->n = x * yBlocks;

	return outList;

}

void GenerateImage(ImageList* structure, ImageList* cache, int x, dim3 resDim,dim3 finalImageSize, Mat *finalImage, bool grayscale, void(*progressCallback)(int, int)) {
	int y = structure->n / x;

	dim3 blockQuant(x, y);
	//dim3 finalImageSize(x * res, y * res);

	//dim3 resDim(res, res);

	ImageData *devData;

	hipMalloc<ImageData>(&devData, sizeof(ImageData) * structure->n);
	hipMemcpy(devData, structure->image, sizeof(ImageData) * structure->n, hipMemcpyHostToDevice);

	//Mat finalImage(x, y, CV_8UC3);
	unsigned char* dFinalImage;

	int sizeFinal = finalImage->rows * finalImage->step;

	hipMalloc<unsigned char>(&dFinalImage, sizeFinal);

	unsigned char* dImageAux;

	dim3 blockKernel(finalImageSize.x / MAX, finalImageSize.y / MAX);
	blockKernel.x++;
	blockKernel.y++;
	dim3 threads(MAX, MAX);	

	set<int> usedImage;

	for (int i = 0; i < structure->n; i++) {
		if (usedImage.find(structure->image[i].hex) != usedImage.end()) {
			continue;
		}
		Mat imgAux = imread(structure->image[i].name);
		int size = imgAux.step * imgAux.rows;

		hipMalloc<unsigned char>(&dImageAux, size);
		hipMemcpy(dImageAux, imgAux.ptr(), size, hipMemcpyHostToDevice);

		dim3 dimAux(imgAux.cols, imgAux.rows);

		FillImageKernel<<<blockKernel, threads>>>(dFinalImage, finalImage->step, resDim, devData, blockQuant, dimAux, dImageAux, imgAux.step, structure->image[i].hex);

		hipDeviceSynchronize();

		hipFree(dImageAux);

		usedImage.insert(structure->image[i].hex);

		if (i % 10 == 0) {
			(*progressCallback)(i, structure->n);
		}

	}

	(*progressCallback)(100, 100);

	if (grayscale) {
		ToGrayScaleKernel<<<blockKernel, threads>>>(dFinalImage, finalImage->step, finalImageSize);
	}

	hipMemcpy(finalImage->ptr(), dFinalImage, sizeFinal, hipMemcpyDeviceToHost);

	return;
}

/*void bestImageTest() {
	clock_t begin = clock();

	Mat image = imread("D:\\igora\\Downloads\\Screenshot_20210318-082317.jpg");
	unsigned char* dImage;

	int size = image.rows * image.step;

	hipMalloc<unsigned char>(&dImage, size);
	hipMemcpy(dImage, image.ptr(), size, hipMemcpyHostToDevice);

	dim3 block(200, 200);
	
	int blockWidth = image.cols / block.x;
	int blockHeight = image.rows / block.y;

	//ImageList* imgList = processImage("D:\\igora\\Documents\\Code\\Photomosaic\\images");
	//saveCache(imgList);
	ImageList* imgList = readCache();
	ImageData* imgData;

	hipMalloc<ImageData>(&imgData, sizeof(ImageData) * imgList->n);
	hipMemcpy(imgData, imgList->image, sizeof(ImageData) * imgList->n, hipMemcpyHostToDevice);

	ImageData* out;
	ImageData* outDevData;

	hipMalloc<ImageData>(&outDevData, sizeof(ImageData) * block.x * block.y);

	out = (ImageData*)malloc(sizeof(ImageData) * block.x * block.y);

	printf("calculando valores...\n");

	AvrgKernel<<<block, 1>>>(dImage, image.step, image.cols, image.rows, imgData, imgList->n, outDevData);

	hipDeviceSynchronize();

	hipMemcpy(out, outDevData, sizeof(ImageData) * block.x * block.y, hipMemcpyDeviceToHost);

	//------------------------------------------------------
	//Leitura das imagens e distribuição

	int quantBlock = block.x * block.y;
	unsigned char** imgArray;
	imgArray = (unsigned char**)malloc(sizeof(char*) * quantBlock);
	
	set<int> usedImg;

	dim3 outSize(50, 50);

	dim3 outImageSize(outSize.x * block.x, outSize.y * block.y);

	Mat outImg(outImageSize.y, outImageSize.x, CV_8UC3);
	unsigned char* outDev;
	int outImgSize = outImg.step * outImg.rows;

	hipMalloc<unsigned char>(&outDev, outImgSize);

	printf("Preenchendo a imagem...\n");

	for (int i = 0; i < quantBlock; i++) {
		if (usedImg.find(out[i].hex) == usedImg.end()) {
			Mat imgAux = imread(out[i].name);
			int size = imgAux.rows * imgAux.step;

			hipMalloc<unsigned char>(&imgArray[i], size);
			hipMemcpy(imgArray[i], imgAux.ptr(), size, hipMemcpyHostToDevice);

			dim3 blockImgSize(imgAux.cols, imgAux.rows);

			dim3 threads(MAX, MAX);

			dim3 blockKernel(outImageSize.x / MAX, outImageSize.y / MAX);

			blockKernel.x++;
			blockKernel.y++;

			FillImageKernel << <blockKernel, threads>> > (outDev, outImg.step, outSize, outDevData, block, blockImgSize, imgArray[i], imgAux.step, out[i].hex);

			usedImg.insert(out[i].hex);
		}

			
	}


	hipDeviceSynchronize();

	printf("salvando imagem...\n");

	hipMemcpy(outImg.ptr(), outDev, outImgSize, hipMemcpyDeviceToHost);

	
	namedWindow("vai");
	imshow("vai", outImg);
	

	imwrite("D:\\igora\\Pictures\\PhotoCuda\\teste.jpg", outImg);

	clock_t end = clock();

	double timeSpent = (double)(end - begin) / CLOCKS_PER_SEC;

	printf("tempo gasto: %.2fs\n", timeSpent);

	waitKey();
} 
*/